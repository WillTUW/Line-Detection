#include "hip/hip_runtime.h"
#include "acc_update.cuh"

//Can you use functions in cuda? Like round?
__global__ void GPU_UpdateAccumulator(int i, int j, int numrho, short* adata, int* max_val, int* max_n)
{
	// update accumulator, find the most probable line
	//for (int n = 0; n < NUM_ANGLE; n++, adata += numrho)
	int n = threadIdx.x;
	if (n >= NUM_ANGLE)
	{
		//wot
		return; //No no you right we want to use return to get out of this func :thumbsup:
	}

	__shared__ int smax_val[NUM_ANGLE];
	__shared__ int smax_n[NUM_ANGLE];

	int r = round(j * hough_cos(n) + i * hough_sin(n)) + ((numrho - 1) / 2);

	adata[r + (n * numrho)] += 1;
	int val = adata[r + (n * numrho)];

	smax_val[n] = val;
	smax_n[n] = n;
	__syncthreads();

	for (int s = 1; s < NUM_ANGLE; s *= 2)
	{
		int index = (2 * s) * n; // Next
		if (index < NUM_ANGLE)
		{
			if (smax_val[index + s] > smax_val[index])
			{
				smax_val[index] = smax_val[index + s];
				smax_n[index] = smax_n[index + s];
			}
		}

		__syncthreads();
	}

	if (n == 0)
	{
		*max_val = smax_val[0];
		*max_n = smax_n[0];
	}
}

void UpdateAccumulator(int i, int j, int numrho, short* dev_adata, int* dev_max_val, int* dev_max_n, 
	short* adata, int *max_val, int *max_n, hipEvent_t cuEvent, hipStream_t stream1, hipStream_t stream2)
{
	int host_max_val[1];
	int host_max_n[1];

	// Copy input vectors from host memory to GPU buffers.

	// old mem copy
	// hipMemcpy(dev_adata, adata, NUM_ANGLE * numrho * sizeof(short), hipMemcpyHostToDevice);

	//*
	// Async mem copy
	hipMemcpyAsync(dev_adata, adata, NUM_ANGLE * numrho * sizeof(short), hipMemcpyHostToDevice, stream1);

	// sync point
	hipEventRecord(cuEvent, stream1); // record event
	hipStreamWaitEvent(stream2, cuEvent, 0); // wait for event in stream1
	//*/

	GPU_UpdateAccumulator <<< 1, NUM_ANGLE, 1, stream2>>> (i, j, numrho, dev_adata, dev_max_val, dev_max_n);

	/*
	// old mem cpy
	hipDeviceSynchronize();
	
	hipMemcpy(adata, dev_adata, NUM_ANGLE * numrho * sizeof(short), hipMemcpyDeviceToHost);
	hipMemcpy(host_max_val, dev_max_val, 1 * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(host_max_n, dev_max_n, 1 * sizeof(int), hipMemcpyDeviceToHost);
	//*/

	//*
	// Async mem copy
	hipMemcpyAsync(adata, dev_adata, NUM_ANGLE * numrho * sizeof(short), hipMemcpyDeviceToHost, stream1);
	hipMemcpyAsync(host_max_val, dev_max_val, 1 * sizeof(int), hipMemcpyDeviceToHost, stream1);
	hipMemcpyAsync(host_max_n, dev_max_n, 1 * sizeof(int), hipMemcpyDeviceToHost, stream1);
	//*/

	*max_val = host_max_val[0];
	*max_n = host_max_n[0];
}
