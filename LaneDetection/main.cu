#include "hip/hip_runtime.h"
/// <summary>
/// Taken from OpenCV implementation of Hough Transform. 
/// Reworked certain functions to work with cuda. Optimizeds using
/// the following strats: 
/// </summary>
#include <opencv2/opencv.hpp>
#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <iostream>
#include <stdio.h>
#include <stdarg.h>
#include <vector>
#include <algorithm>
#include <list>

#define SHIFT 16
#define M_PI 3.14159265359
#define M_THETA (M_PI / 180)
#define RHO 1.0
#define IRHO (1 / RHO)
#define hough_cos(x) (cos(x * M_THETA) * IRHO)
#define hough_sin(x) (sin(x * M_THETA) * IRHO)
#define NUM_ANGLE 180

//Can you use functions in cuda? Like round?
__global__ void GPU_UpdateAccumulator(int i, int j, int numrho, int* adata, int* max_val, int* max_n)
{
	// update accumulator, find the most probable line
	//for (int n = 0; n < NUM_ANGLE; n++, adata += numrho)
	int n = threadIdx.x;
	if (n >= NUM_ANGLE)
	{
		//wot
		return; //No no you right we want to use return to get out of this func :thumbsup:
	}

	__shared__ int smax_val[NUM_ANGLE];
	__shared__ int smax_n[NUM_ANGLE];

	int r = round(j * hough_cos(n) + i * hough_sin(n));
	r += (numrho - 1) / 2;

	int val = ++adata[r + (n * numrho)]; //Affects this line

	smax_val[n] = val;
	smax_n[n] = n;
	__syncthreads();
	
	for (int s = 1; s < NUM_ANGLE; s *= 2)
	{
		int index = (2 * s) * n; // Next
		if (index < NUM_ANGLE) 
		{
			if (smax_val[index + s] > smax_val[index])
			{
				smax_val[index] = smax_val[index + s];
				smax_n[index] = smax_n[index + s];
			}
		}

		__syncthreads();
	}

	if (n == 0)
	{
		*max_val = smax_val[0];
		*max_n = smax_n[0];
	}
}

void UpdateAccumulator(int i, int j, int numrho, int* adata, int *max_val, int *max_n)
{
	int host_max_val[1];
	int host_max_n[1];

	int* dev_adata;
	int* dev_max_val;
	int* dev_max_n;

	hipMalloc((void**)&dev_adata, NUM_ANGLE * numrho * sizeof(int));
	hipMalloc((void**)&dev_max_val, 1 * sizeof(int));
	hipMalloc((void**)&dev_max_n, 1 * sizeof(int));

	// Copy input vectors from host memory to GPU buffers.
	hipMemcpy(dev_adata, adata, NUM_ANGLE * numrho * sizeof(int), hipMemcpyHostToDevice);

	GPU_UpdateAccumulator <<< 1, NUM_ANGLE >>> (i, j, numrho, dev_adata, dev_max_val, dev_max_n);

	hipDeviceSynchronize();

	hipMemcpy(host_max_val, dev_max_val, 1 * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(host_max_n, dev_max_n, 1 * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_adata);
	hipFree(dev_max_val);
	hipFree(dev_max_n);

	*max_val = host_max_val[0];
	*max_n = host_max_n[0];
}

void HoughLinesProbabilistic(cv::Mat& image, int threshold, int lineLength, int lineGap, std::vector<cv::Vec4i>& lines, int linesMax)
{
	cv::Point pt;
	cv::RNG rng((uint64)-1);

	CV_Assert(image.type() == CV_8UC1);

	int width = image.cols;
	int height = image.rows;

	int numrho = cvRound(((width + height) * 2 + 1) / RHO);

	cv::Mat accum = cv::Mat::zeros(NUM_ANGLE, numrho, CV_32SC1);
	cv::Mat mask(height, width, CV_8UC1);

	uchar* mdata0 = mask.ptr();
	std::vector<cv::Point> nzloc;

	// stage 1. collect non-zero image points
	for (pt.y = 0; pt.y < height; pt.y++)
	{
		const uchar* data = image.ptr(pt.y);
		uchar* mdata = mask.ptr(pt.y);
		for (pt.x = 0; pt.x < width; pt.x++)
		{
			if (data[pt.x])
			{
				mdata[pt.x] = (uchar)1;
				nzloc.push_back(pt);
			}
			else
			{
				mdata[pt.x] = 0;
			}
		}
	}

	int count = (int)nzloc.size();

	// stage 2. process all the points in random order
	for (; count > 0; count--)
	{
		// choose random point out of the remaining ones
		int idx = rng.uniform(0, count);
		int max_val = threshold - 1;
		int max_n = 0;
		cv::Point point = nzloc[idx];
		cv::Point line_end[2];
		float a, b;
		int* adata = accum.ptr<int>();
		int i = point.y, j = point.x;
		int k, x0, y0, dx0, dy0;
		bool xflag, good_line;

		// "remove" it by overriding it with the last element
		nzloc[idx] = nzloc[count - 1];

		// check if it has been excluded already (i.e. belongs to some other line)
		if (!mdata0[i*width + j])
			continue;

		// update accumulator, find the most probable line
		UpdateAccumulator(i, j, numrho, adata, &max_val, &max_n);

		//for (int n = 0; n < NUM_ANGLE; n++, adata += numrho)
		//{
		//	int r = cvRound(j * hough_cos(n) + i * hough_sin(n));
		//	r += (numrho - 1) / 2;
		//	int val = ++adata[r];
		//	if (max_val < val)
		//	{
		//		max_val = val;
		//		max_n = n;
		//	}
		//}

		// if it is too "weak" candidate, continue with another point
		if (max_val < threshold)
			continue;

		// from the current point walk in each direction
		// along the found line and extract the line segment
		a = -sin(max_n * M_THETA) * IRHO;
		b = cos(max_n * M_THETA) * IRHO;
		x0 = j;
		y0 = i;
		if (fabs(a) > fabs(b))
		{
			xflag = true;
			dx0 = a > 0 ? 1 : -1;
			dy0 = cvRound(b*(1 << SHIFT) / fabs(a));
			y0 = (y0 << SHIFT) + (1 << (SHIFT - 1));
		}
		else
		{
			xflag = false;
			dy0 = b > 0 ? 1 : -1;
			dx0 = cvRound(a*(1 << SHIFT) / fabs(b));
			x0 = (x0 << SHIFT) + (1 << (SHIFT - 1));
		}

		for (k = 0; k < 2; k++)
		{
			int gap = 0, x = x0, y = y0, dx = dx0, dy = dy0;

			if (k > 0)
				dx = -dx, dy = -dy;

			// walk along the line using fixed-point arithmetic,
			// stop at the image border or in case of too big gap
			for (;; x += dx, y += dy)
			{
				uchar* mdata;
				int i1, j1;

				if (xflag)
				{
					j1 = x;
					i1 = y >> SHIFT;
				}
				else
				{
					j1 = x >> SHIFT;
					i1 = y;
				}

				if (j1 < 0 || j1 >= width || i1 < 0 || i1 >= height)
					break;

				mdata = mdata0 + i1 * width + j1;

				// for each non-zero point:
				//    update line end,
				//    clear the mask element
				//    reset the gap
				if (*mdata)
				{
					gap = 0;
					line_end[k].y = i1;
					line_end[k].x = j1;
				}
				else if (++gap > lineGap)
					break;
			}
		}

		good_line = std::abs(line_end[1].x - line_end[0].x) >= lineLength ||
			std::abs(line_end[1].y - line_end[0].y) >= lineLength;

		for (k = 0; k < 2; k++)
		{
			int x = x0, y = y0, dx = dx0, dy = dy0;
			if (k > 0)
			{
				dx = -dx, dy = -dy;
			}

			// walk along the line using fixed-point arithmetic,
			// stop at the image border or in case of too big gap
			for (;; x += dx, y += dy)
			{
				uchar* mdata;
				int i1, j1;

				if (xflag)
				{
					j1 = x;
					i1 = y >> SHIFT;
				}
				else
				{
					j1 = x >> SHIFT;
					i1 = y;
				}

				mdata = mdata0 + i1 * width + j1;

				// for each non-zero point:
				//    update line end,
				//    clear the mask element
				//    reset the gap
				if (*mdata)
				{
					if (good_line)
					{
						adata = accum.ptr<int>();
						for (int n = 0; n < NUM_ANGLE; n++, adata += numrho)
						{
							int r = cvRound(j1 * hough_cos(n) + i1 * hough_sin(n));
							r += (numrho - 1) / 2;
							adata[r]--;
						}
					}
					*mdata = 0;
				}

				if (i1 == line_end[k].y && j1 == line_end[k].x)
					break;
			}

			if (good_line)
			{
				cv::Vec4i lr(line_end[0].x, line_end[0].y, line_end[1].x, line_end[1].y);
				lines.push_back(lr);
				if ((int)lines.size() >= linesMax)
					return;
			}
		}
	}
}

int main()
{
	const int ddepth = CV_16S;
	const int ksize = 3;

	cv::Mat srcImage = cv::imread("test.png");
	if (srcImage.empty())
	{
		return EXIT_FAILURE;
	}

	// Remove noise by blurring with a Gaussian filter ( kernel size = 3 )
	cv::Mat srcBlurred;
	GaussianBlur(srcImage, srcBlurred, cv::Size(ksize, ksize), 0, 0, cv::BORDER_DEFAULT);

	// Convert the image to grayscale
	cv::Mat srcGray;
	cvtColor(srcBlurred, srcGray, cv::COLOR_BGR2GRAY);

	// Run sobel edge detection
	cv::Mat grad_x, grad_y;
	cv::Sobel(srcGray, grad_x, ddepth, 1, 0, ksize, cv::BORDER_DEFAULT);
	cv::Sobel(srcGray, grad_y, ddepth, 0, 1, ksize, cv::BORDER_DEFAULT);

	// Run canny edge detection
	cv::Mat canny;
	cv::Canny(grad_x, grad_y, canny, 100, 150);
	cv::imwrite("canny.png", canny);

	// Run probabilistic hough line detection
	std::vector<cv::Vec4i> lines;
	HoughLinesProbabilistic(canny, 80, 200, 10, lines, 10);

	// Draw lines detected 
	for (int k = 0; k < lines.size(); k++)
	{
		cv::line(srcImage, cv::Point(lines[k][0], lines[k][1]), cv::Point(lines[k][2], lines[k][3]), cv::Scalar(0, 0, 255), 3, 8);
	}

	// Output image
	cv::imwrite("detected.png", srcImage);
}